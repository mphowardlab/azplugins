#include "hip/hip_runtime.h"
// Copyright (c) 2018-2020, Michael P. Howard
// Copyright (c) 2021, Auburn University
// This file is part of the azplugins project, released under the Modified BSD License.

/*!
 * \file ImplicitDropletEvaporatorGPU.cu
 * \brief Definition of kernel drivers and kernels for ImplicitDropletEvaporatorGPU
 */

#include "ImplicitDropletEvaporatorGPU.cuh"

namespace azplugins
{
namespace gpu
{
namespace kernel
{

/*!
 * \param d_force Particle forces
 * \param d_virial Particle virial
 * \param d_pos Particle positions
 * \param d_params Per-type parameters
 * \param interf_origin Position of interface origin
 * \param N Number of particles
 * \param ntypes Number of types
 *
 * Using one thread per particle, the force of the harmonic potential is computed
 * per-particle. The per-particle-type parameters are cached into shared memory.
 * This method does not compute the virial.
 *
 */
__global__ void compute_implicit_evap_droplet_force(Scalar4 *d_force,
                                                    Scalar *d_virial,
                                                    const Scalar4 *d_pos,
                                                    const Scalar4 *d_params,
                                                    const Scalar interf_origin,
                                                    const unsigned int N,
                                                    const unsigned int ntypes)
    {
    // load per-type parameters into shared memory
    extern __shared__ Scalar4 s_params[];
    for (unsigned int cur_offset = 0; cur_offset < ntypes; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < ntypes)
            {
            s_params[cur_offset + threadIdx.x] = d_params[cur_offset + threadIdx.x];
            }
        }
    __syncthreads();

    // one thread per particle
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    const Scalar4 postype_i = d_pos[idx];
    const Scalar3 pos_i = make_scalar3(postype_i.x, postype_i.y, postype_i.z);
    const unsigned int type_i = __scalar_as_int(postype_i.w);

    const Scalar4 params = s_params[type_i];
    const Scalar k = params.x;
    const Scalar offset = params.y;
    const Scalar g = params.z;
    const Scalar cutoff = params.w;
    // exit if interaction is off
    if (cutoff < Scalar(0.0)) return;

    // get distances and direction of force
    const Scalar r_i = fast::sqrt(dot(pos_i,pos_i));
    const Scalar dr = r_i - (interf_origin + offset);
    if (!(r_i > Scalar(0.0)) || dr < Scalar(0.0)) return;
    const Scalar3 rhat = pos_i/r_i;

    Scalar3 f;
    Scalar e;
    if (dr < cutoff) // harmonic
        {
        f = -k * dr * rhat;
        e = Scalar(0.5) * k * (dr * dr); // (k/2) dr^2
        }
    else // linear
        {
        f = -g * rhat;
        e = Scalar(0.5) * k * cutoff * cutoff + g * (dr - cutoff);
        }

    d_force[idx] = make_scalar4(f.x, f.y, f.z, e);
    }
} // end namespace kernel

/*!
 * \param d_force Particle forces
 * \param d_virial Particle virial
 * \param d_pos Particle positions
 * \param d_params Per-type parameters
 * \param interf_origin Position of interface origin
 * \param N Number of particles
 * \param ntypes Number of types
 * \param block_size Number of threads per block
 *
 * This kernel driver is a wrapper around kernel::compute_implicit_evap_force.
 * The forces and virial are both set to zero before calculation.
 */
hipError_t compute_implicit_evap_droplet_force(Scalar4 *d_force,
                                                Scalar *d_virial,
                                                const Scalar4 *d_pos,
                                                const Scalar4 *d_params,
                                                const Scalar interf_origin,
                                                const unsigned int N,
                                                const unsigned int ntypes,
                                                const unsigned int block_size)
    {
    // zero the force and virial datasets before launch
    hipMemset(d_force, 0, sizeof(Scalar4)*N);
    hipMemset(d_virial, 0, 6*sizeof(Scalar)*N);

    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))kernel::compute_implicit_evap_droplet_force);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);
    unsigned int shared_size = sizeof(Scalar4) * ntypes;

    dim3 grid(N / run_block_size + 1);
    kernel::compute_implicit_evap_droplet_force<<<grid, run_block_size, shared_size>>>(d_force,
                                                                                       d_virial,
                                                                                       d_pos,
                                                                                       d_params,
                                                                                       interf_origin,
                                                                                       N,
                                                                                       ntypes);
    return hipSuccess;
    }

} // end namespace gpu
} // end namespace azplugins

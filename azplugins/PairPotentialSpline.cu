// Copyright (c) 2018-2020, Michael P. Howard
// Copyright (c) 2021, Auburn University
// This file is part of the azplugins project, released under the Modified BSD License.

#include "PairPotentials.cuh"

namespace azplugins
{
namespace gpu
{

//! Kernel driver for spline pair potential
template hipError_t compute_pair_potential<azplugins::detail::PairEvaluatorSpline>
    (const pair_args_t& pair_args,
     const typename azplugins::detail::PairEvaluatorSpline::param_type *d_params);

} // end namespace gpu
} // end namespace azplugins

// Copyright (c) 2018-2020, Michael P. Howard
// Copyright (c) 2021, Auburn University
// This file is part of the azplugins project, released under the Modified BSD License.

#include "PairPotentials.cuh"

namespace azplugins
{
namespace gpu
{

//! Kernel driver for LJ 12-4 pair potential
template hipError_t compute_pair_potential<azplugins::detail::PairEvaluatorLJ124>
     (const pair_args_t& pair_args,
     const typename azplugins::detail::PairEvaluatorLJ124::param_type *d_params);

} // end namespace gpu
} // end namespace azplugins

#include "hip/hip_runtime.h"
// Copyright (c) 2018-2020, Michael P. Howard
// Copyright (c) 2021, Auburn University
// This file is part of the azplugins project, released under the Modified BSD License.

/*!
 * \file TwoStepLangevinFlowGPU.cu
 * \brief Definition of kernel drivers and kernels for TwoStepLangevinFlowGPU
 */

#include "TwoStepLangevinFlowGPU.cuh"
#include "FlowFields.h"

namespace azplugins
{
namespace gpu
{
namespace kernel
{
__global__ void langevin_flow_step1(Scalar4 *d_pos,
                                    int3 *d_image,
                                    Scalar4 *d_vel,
                                    const Scalar3 *d_accel,
                                    const unsigned int *d_group,
                                    const BoxDim box,
                                    const unsigned int N,
                                    const Scalar dt)
    {
    const unsigned int grp_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (grp_idx >= N) return;

    const unsigned int idx = d_group[grp_idx];

    // position
    const Scalar4 postype = d_pos[idx];
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    unsigned int type = __scalar_as_int(postype.w);

    // velocity
    const Scalar4 velmass = d_vel[idx];
    Scalar3 vel = make_scalar3(velmass.x, velmass.y, velmass.z);
    Scalar mass = velmass.w;

    // acceleration
    const Scalar3 accel = d_accel[idx];

    // update position and wrap
    pos += (vel + Scalar(0.5) * dt * accel) * dt;
    int3 image = d_image[idx];
    box.wrap(pos,image);

    // update velocity
    vel += Scalar(0.5) * dt * accel;

    d_pos[idx] = make_scalar4(pos.x, pos.y, pos.z, __int_as_scalar(type));
    d_vel[idx] = make_scalar4(vel.x, vel.y, vel.z, mass);
    d_image[idx] = image;
    }
} // end namespace kernel

hipError_t langevin_flow_step1(Scalar4 *d_pos,
                                int3 *d_image,
                                Scalar4 *d_vel,
                                const Scalar3 *d_accel,
                                const unsigned int *d_group,
                                const BoxDim& box,
                                const unsigned int N,
                                const Scalar dt,
                                const unsigned int block_size)
    {
    if (N == 0) return hipSuccess;

    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))kernel::langevin_flow_step1);
        max_block_size = attr.maxThreadsPerBlock;
        }

    const int run_block_size = min(block_size, max_block_size);
    kernel::langevin_flow_step1<<<N/run_block_size+1, run_block_size>>>(d_pos,
                                                                        d_image,
                                                                        d_vel,
                                                                        d_accel,
                                                                        d_group,
                                                                        box,
                                                                        N,
                                                                        dt);
    return hipSuccess;
    }

//! Explicit instantiation of ConstantFlow integrator
template hipError_t langevin_flow_step2<azplugins::ConstantFlow>(Scalar4 *d_vel,
                                                                  Scalar3 *d_accel,
                                                                  const Scalar4 *d_pos,
                                                                  const Scalar4 *d_net_force,
                                                                  const unsigned int *d_tag,
                                                                  const unsigned int *d_group,
                                                                  const Scalar *d_diameter,
                                                                  const Scalar lambda,
                                                                  const Scalar *d_gamma,
                                                                  const unsigned int ntypes,
                                                                  const azplugins::ConstantFlow& flow_field,
                                                                  const unsigned int N,
                                                                  const Scalar dt,
                                                                  const Scalar T,
                                                                  const unsigned int timestep,
                                                                  const unsigned int seed,
                                                                  bool noiseless,
                                                                  bool use_lambda,
                                                                  const unsigned int block_size);
//! Explicit instantiation of ParabolicFlow integrator
template hipError_t langevin_flow_step2<azplugins::ParabolicFlow>(Scalar4 *d_vel,
                                                                   Scalar3 *d_accel,
                                                                   const Scalar4 *d_pos,
                                                                   const Scalar4 *d_net_force,
                                                                   const unsigned int *d_tag,
                                                                   const unsigned int *d_group,
                                                                   const Scalar *d_diameter,
                                                                   const Scalar lambda,
                                                                   const Scalar *d_gamma,
                                                                   const unsigned int ntypes,
                                                                   const azplugins::ParabolicFlow& flow_field,
                                                                   const unsigned int N,
                                                                   const Scalar dt,
                                                                   const Scalar T,
                                                                   const unsigned int timestep,
                                                                   const unsigned int seed,
                                                                   bool noiseless,
                                                                   bool use_lambda,
                                                                   const unsigned int block_size);

//! Explicit instantiation of QuiescentFluid integrator
template hipError_t langevin_flow_step2<azplugins::QuiescentFluid>(Scalar4 *d_vel,
                                                                    Scalar3 *d_accel,
                                                                    const Scalar4 *d_pos,
                                                                    const Scalar4 *d_net_force,
                                                                    const unsigned int *d_tag,
                                                                    const unsigned int *d_group,
                                                                    const Scalar *d_diameter,
                                                                    const Scalar lambda,
                                                                    const Scalar *d_gamma,
                                                                    const unsigned int ntypes,
                                                                    const azplugins::QuiescentFluid& flow_field,
                                                                    const unsigned int N,
                                                                    const Scalar dt,
                                                                    const Scalar T,
                                                                    const unsigned int timestep,
                                                                    const unsigned int seed,
                                                                    bool noiseless,
                                                                    bool use_lambda,
                                                                    const unsigned int block_size);

} // end namespace gpu
} // end namespace azplugins

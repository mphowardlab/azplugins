#include "hip/hip_runtime.h"
// Copyright (c) 2018-2020, Michael P. Howard
// Copyright (c) 2021-2022, Auburn University
// This file is part of the azplugins project, released under the Modified BSD License.

// Maintainer: astatt

/*!
 * \file SinusoidalExpansionConstrictionFillerGPU.cu
 * \brief Defines GPU functions and kernels used by azplugins::gpu::SinusoidalExpansionConstrictionFillerGPU
 */

#include "SinusoidalExpansionConstrictionFillerGPU.cuh"
#include "hoomd/RandomNumbers.h"
#include "RNGIdentifiers.h"
#include "hoomd/mpcd/ParticleDataUtilities.h"

namespace azplugins
{

namespace gpu
{

namespace kernel
{
/*!
 * \param d_pos Particle positions
 * \param d_vel Particle velocities
 * \param d_tag Particle tags
 * \param geom geometry to fill
 * \param m_pi_period_div_L
 * \param m_amplitude
 * \param box Local simulation box
 * \param type Type of fill particles
 * \param N_lo Number of particles to fill in lower region
 * \param N_hi Number of particles to fill in upper region
 * \param first_tag First tag of filled particles
 * \param first_idx First (local) particle index of filled particles
 * \param vel_factor Scale factor for uniform normal velocities consistent with particle mass / temperature
 * \param timestep Current timestep
 * \param seed User seed to PRNG for drawing velocities
 *
 * \b Implementation:
 *
 * Using one thread per particle (in both slabs), the thread is assigned to fill either the lower
 * or upper region.  A random position is drawn within a cuboid, which is then shifted in z using the wall equation.
 * The thread index is translated into a particle tag and local particle index.
 */
__global__ void sin_expansion_constriction_draw_particles(Scalar4 *d_pos,
                                                          Scalar4 *d_vel,
                                                          unsigned int *d_tag,
                                                          const azplugins::detail::SinusoidalExpansionConstriction geom,
                                                          const Scalar m_pi_period_div_L,
                                                          const Scalar m_amplitude,
                                                          const Scalar m_H_narrow,
                                                          const Scalar m_thickness,
                                                          const BoxDim box,
                                                          const unsigned int type,
                                                          const unsigned int N_fill,
                                                          const unsigned int first_tag,
                                                          const unsigned int first_idx,
                                                          const Scalar vel_factor,
                                                          const unsigned int timestep,
                                                          const unsigned int seed)
    {
    // one thread per particle
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_fill)
        return;
    Scalar3 lo = box.getLo();
    Scalar3 hi = box.getHi();
    const unsigned int N_half = 0.5*N_fill;

    // particle tag and index
    const unsigned int tag = first_tag + idx;
    const unsigned int pidx = first_idx + idx;
    d_tag[pidx] = tag;

    // initialize random number generator for positions and velocity
    hoomd::RandomGenerator rng(RNGIdentifier::SinusoidalExpansionConstrictionFiller, seed, tag, timestep);
    signed char sign = (idx >= N_half) - (idx < N_half); // bottom -1 or top +1

    Scalar x = hoomd::UniformDistribution<Scalar>(lo.x, hi.x)(rng);
    Scalar y = hoomd::UniformDistribution<Scalar>(lo.y, hi.y)(rng);
    Scalar z = hoomd::UniformDistribution<Scalar>(0, sign*m_thickness)(rng);
    z += sign*(m_amplitude*fast::cos(x*m_pi_period_div_L)+m_amplitude + m_H_narrow );

    d_pos[pidx] = make_scalar4(x, y, z, __int_as_scalar(type));

    hoomd::NormalDistribution<Scalar> gen(vel_factor, 0.0);
    Scalar3 vel;
    gen(vel.x, vel.y, rng);
    vel.z = gen(rng);

    d_vel[pidx] = make_scalar4(vel.x,
                               vel.y,
                               vel.z,
                               __int_as_scalar(mpcd::detail::NO_CELL));
    }
} // end namespace kernel

/*!
 * \param d_pos Particle positions
 * \param d_vel Particle velocities
 * \param d_tag Particle tags
 * \param geom Slit geometry to fill
 * \param z_min Lower bound to lower fill region
 * \param z_max Upper bound to upper fill region
 * \param box Local simulation box
 * \param mass Mass of fill particles
 * \param type Type of fill particles
 * \param N_lo Number of particles to fill in lower region
 * \param N_hi Number of particles to fill in upper region
 * \param first_tag First tag of filled particles
 * \param first_idx First (local) particle index of filled particles
 * \param kT Temperature for fill particles
 * \param timestep Current timestep
 * \param seed User seed to PRNG for drawing velocities
 * \param block_size Number of threads per block
 *
 * \sa kernel::sin_expansion_constriction_draw_particles
 */
hipError_t sin_expansion_constriction_draw_particles(Scalar4 *d_pos,
                                   Scalar4 *d_vel,
                                   unsigned int *d_tag,
                                   const azplugins::detail::SinusoidalExpansionConstriction& geom,
                                   const Scalar m_pi_period_div_L,
                                   const Scalar m_amplitude,
                                   const Scalar m_H_narrow,
                                   const Scalar m_thickness,
                                   const BoxDim& box,
                                   const Scalar mass,
                                   const unsigned int type,
                                   const unsigned int N_fill,
                                   const unsigned int first_tag,
                                   const unsigned int first_idx,
                                   const Scalar kT,
                                   const unsigned int timestep,
                                   const unsigned int seed,
                                   const unsigned int block_size)
    {
    if (N_fill == 0) return hipSuccess;

    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))kernel::sin_expansion_constriction_draw_particles);
        max_block_size = attr.maxThreadsPerBlock;
        }

    // precompute factor for rescaling the velocities since it is the same for all particles
    const Scalar vel_factor = fast::sqrt(kT / mass);

    unsigned int run_block_size = min(block_size, max_block_size);
    dim3 grid(N_fill / run_block_size + 1);
    kernel::sin_expansion_constriction_draw_particles<<<grid, run_block_size>>>(d_pos,
                                                             d_vel,
                                                             d_tag,
                                                             geom,
                                                             m_pi_period_div_L,
                                                             m_amplitude,
                                                             m_H_narrow,
                                                             m_thickness,
                                                             box,
                                                             type,
                                                             N_fill,
                                                             first_tag,
                                                             first_idx,
                                                             vel_factor,
                                                             timestep,
                                                             seed);

    return hipSuccess;
    }

} // end namespace gpu
} // end namespace azplugins
